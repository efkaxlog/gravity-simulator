
#include <hip/hip_runtime.h>
#include <math.h>

extern "C"
__global__ void getAcceleration(float *bodyX, float *bodyY, float *body2X, float *body2Y, float *mass, float *mass2, float *g, float *dx, float *dy)
                    /*** kernel code ***/
{
    float d = sqrt(pow (bodyX[0] - body2X[0], 2) + pow(bodyY[0] - body2Y[0], 2));
    dx[0] = g[0] * mass[0] / (d*d) * (bodyX[0] - body2X[0]) / d;
    dy[0] = g[0] * mass[0] / (d*d) * (bodyY[0] - body2Y[0]) / d;
}
